#include "hip/hip_runtime.h"
#include <chrono>
#include <cstdio>
#include <fstream>
#include <iostream>
#include <sstream>
#include <thread>

#include "DataReader2.h"
#include "helper.cuh"
#include "nonorm.cuh"
#include "norm.cuh"
#include "utils.h"

int main(int argc, char * argv[]) 
{
  try 
  {
    Config conf(argv[1]);
    conf.print(); // 逐行展示从配置文件中读取的各个参数

    auto lst = LST::load(conf.gets("Input")); // 从Input中读取参数（欠焦值、像散等）
    EulerData euler(conf.gets("Euler_angles_file")); // 读取欧拉角

    auto device = conf.geti("GPU_ID");
    std::printf("Selected device ID: %d\n", device);

    auto first = std::max(0, std::min(conf.geti("First_image"), int(lst.size() - 1)));
    auto last = std::min(conf.geti("Last_image"), std::max(0, int(lst.size())));

    INIT_TIMEIT();

    Templates temp;
    std::printf("Picking templates: %s, ", conf.gets("Picking_templates").c_str());
    TIMEIT(temp = Templates(conf.gets("Picking_templates"), euler.size()));

    std::fstream output(conf.gets("Output"), std::ios::out | std::ios::trunc);
    //std::cout << "Output file name: " << conf.gets("Output") << ".\n";

    if (device != -1) 
    {
      for (auto i = first; i < last; ++i) 
      {
        const auto & entry = lst[i];
        if (conf.geti("Norm_type")) 
        {
          auto image = Image{entry};
          auto params = image.p;
          SearchNorm p(conf, euler, {params.width, params.height}, device);

          TIMEIT(p.work_verbose(temp, image, output); std::printf("Device %d finished in ", device););
        } 
        else 
        {
          SearchNoNorm p(conf, euler, {tile_size, tile_size}, device);
          auto tiles = TileImages{entry};
          TIMEIT(p.work_verbose(temp, tiles, output); std::printf("Device %d finished in ", device););
        }
      }
    } else {
      auto devcount = GetDeviceCount();
      std::printf("Device count: %d\n", devcount);
      auto intervals = work_intervals(first, last, devcount);

      std::vector<std::stringstream> ss(devcount);

      auto worker = [&](int device, std::pair<int, int> interval) {
        INIT_TIMEIT();
        std::stringstream output;
        for (auto i = interval.first; i < interval.second; ++i) {
          const auto& entry = lst[i];
          if (conf.geti("Norm_type")) {
            auto image = Image{entry};
            auto params = image.p;
            SearchNorm p(conf, euler, {params.width, params.height}, device);

            p.work(temp, image, output);
          } else {
            SearchNoNorm p(conf, euler, {tile_size, tile_size}, device);
            auto tiles = TileImages{entry};
            TIMEIT(p.work(temp, tiles, output); std::printf("Device %d finished in ", device););
          }
        }
        ss[device] = std::move(output);
      };

      auto wcount = std::min(devcount, last - first);
      std::vector<std::thread> ts(wcount);
      for (auto dev = 0; dev < wcount; ++dev) {
        ts[dev] = std::thread(worker, dev, intervals[dev]);
      }

      for (auto& t : ts) {
        t.join();
      }

      for (const auto& s : ss) {
        output << s.rdbuf();
      }
    }
  } catch (const std::exception& e) {
    std::cout << e.what() << std::endl;
    std::exit(-1);
  }

  return 0;
}